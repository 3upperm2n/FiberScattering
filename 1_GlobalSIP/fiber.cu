#include "hip/hip_runtime.h"
/*
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h> // getopt
#include <time.h>

#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_math_constants.h>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h> // float4
#include <hip/hip_runtime_api.h> // check error


#define FOUR_PI (4*HIP_PI_F)
#define INV_PI  (1/HIP_PI_F)
#define TK 1 // time kernel
#define DB 0 // debug 


__device__ __constant__ float 
d_atomC[9]={ 2.31000,  1.02000,  1.58860,  0.865000, 
             20.8439,  10.2075, 0.568700,   51.6512,  
             0.2156};

__device__ __constant__ float 
d_atomH[9]={ 0.493002, 0.322912, 0.140191, 0.040810, 
             10.5109,  26.1257, 3.14236,  57.7997,
			 0.003038};

__device__ __constant__ float 
d_atomO[9]={ 3.04850,  2.28680,  1.54630,  0.867000, 
             13.2771,  5.70110, 0.323900, 32.9089,
			 0.2508};

__device__ __constant__ float 
d_atomN[9]={ 12.2126,  3.13220,  2.01250,  1.16630,  
             0.005700, 9.89330, 28.9975,  0.582600, 
			 -11.52};


// global value
char* fname;
float lamda;
float distance;
int   span;
int   nstreams;

float *q;
float4 *formfactor;

// event timing
hipEvent_t start, stop;
float elapsedTime;

// cuda streams
hipStream_t *streams;


void Usage(char *argv0)
{
	const char *help_msg =	
		"\nUsage: %s [options] -f filename\n\n"
		"    -f filename      :file containing atom info\n"		
		"    -l lamda         :angstrom value                 [default=1.033]\n"
		"    -d distance      :specimen to detector distance  [default=300]\n"
		"    -s span          :sampling resolution            [default=2048]\n"
		"    -n nstreams      :number of cuda streams         [default=2]\n";
	fprintf(stderr, help_msg, argv0);
	exit(-1);
}


__global__ void kernel_qr(float *q, 
                          int N, 
						  float inv_lamda, 
						  float inv_distance)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (gid < N)
	{
		q[gid] = FOUR_PI * inv_lamda * sin(0.5 * atan( gid * 0.0732f * inv_distance));
	}
}

__global__ void kernel_prepare(float *q,
                                 int N,
							     float4 *formfactor,
								 float inv_lamda,
								 float inv_distance)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (gid < N)
	{
		float tmp, local_q;
		float fc, fh, fo, fn;

		local_q = FOUR_PI * inv_lamda * sin(0.5 * atan(gid * 0.0732f * inv_distance));
		tmp = -powf(local_q * 0.25 * INV_PI, 2.0);

		// loop unrolling
		fc = d_atomC[0] * expf(d_atomC[4] * tmp) +
			 d_atomC[1] * expf(d_atomC[5] * tmp) +
			 d_atomC[2] * expf(d_atomC[6] * tmp) +
			 d_atomC[3] * expf(d_atomC[7] * tmp) +
			 d_atomC[8];

		fh = d_atomH[0] * expf(d_atomH[4] * tmp) +
			d_atomH[1] * expf(d_atomH[5] * tmp) +
			d_atomH[2] * expf(d_atomH[6] * tmp) +
			d_atomH[3] * expf(d_atomH[7] * tmp) +
			d_atomH[8];

		fo = d_atomO[0] * expf(d_atomO[4] * tmp) +
			d_atomO[1] * expf(d_atomO[5] * tmp) +
			d_atomO[2] * expf(d_atomO[6] * tmp) +
			d_atomO[3] * expf(d_atomO[7] * tmp) +
			d_atomO[8];

		fn = d_atomN[0] * expf(d_atomN[4] * tmp) +
			d_atomN[1] * expf(d_atomN[5] * tmp) +
			d_atomN[2] * expf(d_atomN[6] * tmp) +
			d_atomN[3] * expf(d_atomN[7] * tmp) +
			d_atomN[8];

		formfactor[gid] = make_float4(fc, fh, fo, fn);
		q[gid]          = local_q;
	}
}

__global__ void kernel_qr_factor(float *q,
                                 int N,
							     float4 *formfactor,
								 float inv_lamda,
								 float inv_distance,
								 int offset)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x + offset;

	if (gid < N)
	{
		float tmp, local_q;
		float fc, fh, fo, fn;

		local_q = FOUR_PI * inv_lamda * sin(0.5 * atan(gid * 0.0732f * inv_distance));
		tmp = -powf(local_q * 0.25 * INV_PI, 2.0);

		// loop unrolling
		fc = d_atomC[0] * expf(d_atomC[4] * tmp) +
			 d_atomC[1] * expf(d_atomC[5] * tmp) +
			 d_atomC[2] * expf(d_atomC[6] * tmp) +
			 d_atomC[3] * expf(d_atomC[7] * tmp) +
			 d_atomC[8];

		fh = d_atomH[0] * expf(d_atomH[4] * tmp) +
			d_atomH[1] * expf(d_atomH[5] * tmp) +
			d_atomH[2] * expf(d_atomH[6] * tmp) +
			d_atomH[3] * expf(d_atomH[7] * tmp) +
			d_atomH[8];

		fo = d_atomO[0] * expf(d_atomO[4] * tmp) +
			d_atomO[1] * expf(d_atomO[5] * tmp) +
			d_atomO[2] * expf(d_atomO[6] * tmp) +
			d_atomO[3] * expf(d_atomO[7] * tmp) +
			d_atomO[8];

		fn = d_atomN[0] * expf(d_atomN[4] * tmp) +
			d_atomN[1] * expf(d_atomN[5] * tmp) +
			d_atomN[2] * expf(d_atomN[6] * tmp) +
			d_atomN[3] * expf(d_atomN[7] * tmp) +
			d_atomN[8];

		formfactor[gid + offset] = make_float4(fc, fh, fo, fn);
		q[gid + offset]          = local_q;
	}
}

__global__ void kernel_factor_v1(int N,
							     float4 *formfactor,
								 float inv_lamda,
								 float inv_distance)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (gid < N)
	{
		float tmp;
		float fc, fh, fo, fn;

		tmp = FOUR_PI * inv_lamda * sin(0.5 * atan( gid * 0.0732f * inv_distance));
		tmp = -powf(tmp * 0.25 * INV_PI, 2.0);

		// loop unrolling
		fc = d_atomC[0] * expf(d_atomC[4] * tmp) +
			 d_atomC[1] * expf(d_atomC[5] * tmp) +
			 d_atomC[2] * expf(d_atomC[6] * tmp) +
			 d_atomC[3] * expf(d_atomC[7] * tmp) +
			 d_atomC[8];

		fh = d_atomH[0] * expf(d_atomH[4] * tmp) +
			d_atomH[1] * expf(d_atomH[5] * tmp) +
			d_atomH[2] * expf(d_atomH[6] * tmp) +
			d_atomH[3] * expf(d_atomH[7] * tmp) +
			d_atomH[8];

		fo = d_atomO[0] * expf(d_atomO[4] * tmp) +
			d_atomO[1] * expf(d_atomO[5] * tmp) +
			d_atomO[2] * expf(d_atomO[6] * tmp) +
			d_atomO[3] * expf(d_atomO[7] * tmp) +
			d_atomO[8];

		fn = d_atomN[0] * expf(d_atomN[4] * tmp) +
			d_atomN[1] * expf(d_atomN[5] * tmp) +
			d_atomN[2] * expf(d_atomN[6] * tmp) +
			d_atomN[3] * expf(d_atomN[7] * tmp) +
			d_atomN[8];

		formfactor[gid] = make_float4(fc, fh, fo, fn);
	}
}

__global__ void kernel_factor(float *q, 
                              int N,
							  float4 *formfactor)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	// fixme : use vector instruction
	if (gid < N)
	{
		float tmp;
		float fc, fh, fo, fn;
		tmp = -powf(q[gid] * 0.25 * INV_PI, 2.0);

		// loop unrolling
		fc = d_atomC[0] * expf(d_atomC[4] * tmp) +
			 d_atomC[1] * expf(d_atomC[5] * tmp) +
			 d_atomC[2] * expf(d_atomC[6] * tmp) +
			 d_atomC[3] * expf(d_atomC[7] * tmp) +
			 d_atomC[8];

		fh = d_atomH[0] * expf(d_atomH[4] * tmp) +
			d_atomH[1] * expf(d_atomH[5] * tmp) +
			d_atomH[2] * expf(d_atomH[6] * tmp) +
			d_atomH[3] * expf(d_atomH[7] * tmp) +
			d_atomH[8];

		fo = d_atomO[0] * expf(d_atomO[4] * tmp) +
			d_atomO[1] * expf(d_atomO[5] * tmp) +
			d_atomO[2] * expf(d_atomO[6] * tmp) +
			d_atomO[3] * expf(d_atomO[7] * tmp) +
			d_atomO[8];

		fn = d_atomN[0] * expf(d_atomN[4] * tmp) +
			d_atomN[1] * expf(d_atomN[5] * tmp) +
			d_atomN[2] * expf(d_atomN[6] * tmp) +
			d_atomN[3] * expf(d_atomN[7] * tmp) +
			d_atomN[8];

		formfactor[gid] = make_float4(fc, fh, fo, fn);
	}
}

void run_qr()
{
	// fixe me : use occupancy api
	dim3 block(256, 1, 1);
	dim3 grid(ceil((float) span / block.x ), 1, 1);

//#if TK
//	hipEventRecord(start, 0);
//#endif


	kernel_qr <<< grid, block >>> (q, span, 1.f/lamda, 1.f/distance);

//#if TK
//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("kernel_qr = %f ms\n", elapsedTime);
//#endif

	
#if DB
	hipDeviceSynchronize();	

	for(int i=0; i<span; i++)
		printf("q[%d] : %f\n", i, q[i]);
#endif
}



void run_factor()
{
	// fixe me : use occupancy api
	dim3 block(256, 1, 1);
	dim3 grid(ceil((float) span / block.x ), 1, 1);

//#if TK
//	hipEventRecord(start, 0);
//#endif

	kernel_factor <<< grid, block >>> (q, span, formfactor);

//#if TK
//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("kernel_factor = %f ms\n", elapsedTime);
//#endif

	
#if DB
	hipDeviceSynchronize();	

	printf("\t\tC\t\tH\t\tO\t\tN\n");

	for(int i = 0; i < span; i++)
	{
		printf("factor[%d] :\t%f\t%f\t%f\t%f\n", i, 
		                                         formfactor[i].x,
		                                         formfactor[i].y,
		                                         formfactor[i].z,
		                                         formfactor[i].w);
	}
#endif



}


int main(int argc, char*argv[])
{
	//-----------------------------------------------------------------------//
	// Read input
	//-----------------------------------------------------------------------//
	lamda = 1.033f; 
	distance = 300.f;
	span = 2048;
	nstreams = 2;

	int opt;
	int fflag = 0;
	extern char   *optarg;  

	// fixme: detect wrong options
	while ( (opt=getopt(argc,argv,"f:l:d:s:n:"))!= EOF) 
	{                    
		switch (opt) {                                                          
			case 'f': 
			         fflag = 1;               // mandatory
			         fname = optarg;                                          
					 break;
			case 'l':
			         lamda = atof(optarg); 
					 break;                                                      
			case 'd': 
			         distance = atof(optarg);
					 break;                                                      
			case 's': 
			         span = atoi(optarg);                             
					 break;                                                      
			case 'n': 
			         nstreams = atoi(optarg);                             
					 break;                                                      
			case '?': 
			         Usage(argv[0]);                                           
					 break;                                                      
			default: 
			         Usage(argv[0]);                                            
					 break;                                                      
		}                                                                       
	}                                                                       

	if (fname == 0) Usage(argv[0]);  

	if (fflag == 0) 
	{
		fprintf(stderr, "%s: missing -f option\n", argv[0]);
        Usage(argv[0]);                                            
	} 

	// check
	std::cout << "file name : " << fname     << std::endl;
	std::cout << "lamda : "     << lamda     << std::endl;
	std::cout << "distance: "   << distance  << std::endl;
	std::cout << "span: "       << span      << std::endl;
	std::cout << "streams: "    << nstreams  << std::endl;

	//-----------------------------------------------------------------------//
	// GPU  
	//-----------------------------------------------------------------------//
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    std::cout << "max texture1d linear: " << deviceProp.maxTexture1DLinear << std::endl;

#if TK
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif




	// um 
	hipMallocManaged((void**)&q,          sizeof(float) * span);
	hipMallocManaged((void**)&formfactor, sizeof(float4) * span);

	// streams
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	for (int i = 0 ; i < nstreams ; i++){
		checkCudaErrors(hipStreamCreate(&(streams[i])));
	}

	// step 1
	// run_qr();

	// step 2
	// run_factor();


#if TK
	hipEventRecord(start, 0);
#endif

	kernel_prepare <<< grid, block >>> (q, span, formfactor, 1.f/lamda, 1.f/distance);


#if TK
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("runtime = %f ms\n", elapsedTime);
#endif


	// release
	hipFree(q);
	hipFree(formfactor);

	for (int i = 0 ; i < nstreams ; i++){
		hipStreamDestroy(streams[i]);
	}
	free(streams);

	hipDeviceReset();



/*

	for(int i=0; i<nstreams; i++)
	{
		checkCudaErrors(hipMemcpyAsync(&h_Iq[i * N],   &d_Iq[i * N],  sizeof(float) * N, hipMemcpyDeviceToHost, streams[i]));
		checkCudaErrors(hipMemcpyAsync(&h_Iqz[i * N], &d_Iqz[i * N],  sizeof(float) * N, hipMemcpyDeviceToHost, streams[i]));
	
	}

	hipDeviceSynchronize();

	for(int i=0; i < N; i++){
		for(int s=1; s<nstreams; s++){
			h_Iq[i]  += h_Iq[i + s * N];	
			h_Iqz[i] += h_Iqz[i + s * N];	
		}
	}


	// release resources

	hipUnbindTexture(crdTex);	


	hipFree(d_q);
	hipFree(d_R);
	hipFree(d_factor);
	hipFree(d_Iq);
	hipFree(d_Iqz);
	hipFree(d_crd);

	hipHostFree(h_Iq);
	hipHostFree(h_Iqz);

#if DB
	free(q);
	free(R);
	free(factor);
#endif

	checkCudaErrors(hipDeviceReset());
*/
	exit (EXIT_SUCCESS);
}
