#include "hip/hip_runtime.h"
/*
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h> // getopt
#include <time.h>

#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_math_constants.h>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h> // float4
#include <hip/hip_runtime_api.h> // check error


#define FOUR_PI (4*HIP_PI_F)
#define INV_PI  (1/HIP_PI_F)
#define TK 1 // time kernel
#define DB 0 // debug 


__device__ __constant__ float 
d_atomC[9]={ 2.31000,  1.02000,  1.58860,  0.865000, 
             20.8439,  10.2075, 0.568700,   51.6512,  
             0.2156};

__device__ __constant__ float 
d_atomH[9]={ 0.493002, 0.322912, 0.140191, 0.040810, 
             10.5109,  26.1257, 3.14236,  57.7997,
			 0.003038};

__device__ __constant__ float 
d_atomO[9]={ 3.04850,  2.28680,  1.54630,  0.867000, 
             13.2771,  5.70110, 0.323900, 32.9089,
			 0.2508};

__device__ __constant__ float 
d_atomN[9]={ 12.2126,  3.13220,  2.01250,  1.16630,  
             0.005700, 9.89330, 28.9975,  0.582600, 
			 -11.52};

__device__ __constant__ char   atom_type_const[60000];

texture<float4, 1, hipReadModeElementType> crdc_tex;
texture<float4, 1, hipReadModeElementType> crdh_tex;
texture<float4, 1, hipReadModeElementType> crdo_tex;

// global value
char     *fname;
float     lamda;
float     distance;
int       span;
int       nstreams;
int       linenum;
int       line_c;
int       line_h;
int       line_o;

// unified memory
float     *q;
float4    *formfactor;
// char      *atom_type;

float4    *crd_c;
float4    *crd_h;
float4    *crd_o;

// event timing
hipEvent_t start, stop;
float elapsedTime;

// cuda streams
hipStream_t *streams;

//std::vector<char>  atom_type;
//std::vector<float4> crd;

void Usage(char *argv0)
{
	const char *help_msg =	
		"\nUsage: %s [options] -f filename\n\n"
		"    -f filename      :file containing atom info\n"		
		"    -l lamda         :angstrom value                 [default=1.033]\n"
		"    -d distance      :specimen to detector distance  [default=300]\n"
		"    -s span          :sampling resolution            [default=2048]\n"
		"    -n nstreams      :number of cuda streams         [default=2]\n";
	fprintf(stderr, help_msg, argv0);
	exit(-1);
}

void readpdb()
{
	// Steps:
	// search the first line start from ATOM
	// the 3rd column, first character is the atom type 
	// 6, 7, 8 column is the x, y, z corordinates

	char line[1000];
	char c1[20];
	char atominfo[20];
	float x, y, z;

	FILE *fp = fopen(fname,"r");
	if(fp == NULL)
		perror("Error opening file!!!\n\n");

	char s;
	linenum = 0;
	line_c  = 0;
	line_h  = 0;
	line_o  = 0;

	while (fgets(line,1000,fp)!=NULL)
	{
		sscanf(line, "%s", c1);
		if(!(strcmp(c1, "ATOM")))
		{
			linenum++;                               // atom list length
			sscanf(line, "%*s %*d %s", atominfo);
			s = atominfo[0];
			if(s == 'C')
				line_c++;
			if(s == 'H')
				line_h++;
			if(s == 'O')
				line_o++;
		}
	}

	rewind(fp);

//	std::cout << linenum << std::endl;
	std::cout << line_c << std::endl;
	std::cout << line_h << std::endl;
	std::cout << line_o << std::endl;

	// unified memory 
	// hipMallocManaged((void**)&atom_type,      sizeof(char) * linenum);
	//hipMallocManaged((void**)&crd,          sizeof(float4) * linenum);

	// unified memory
	hipMallocManaged((void**)&crd_c, sizeof(float4) * line_c);
	hipMallocManaged((void**)&crd_h, sizeof(float4) * line_h);
	hipMallocManaged((void**)&crd_o, sizeof(float4) * line_o);

	int id_c, id_h, id_o;
	id_c = id_h = id_o = 0;

	while (fgets(line,1000,fp)!=NULL)
	{
		sscanf(line, "%s", c1);
		if(!(strcmp(c1, "ATOM")))
		{
			sscanf(line, "%*s %*d %s %*s %*d %f %f %f", atominfo, &x, &y, &z);
			s = atominfo[0];
			if(s == 'C')
			{
				crd_c[id_c] = make_float4(x, y, z, 0.f);
				id_c++;
			}

			if(s == 'H')
			{
				crd_h[id_h] = make_float4(x, y, z, 0.f);
				id_h++;	
			}

			if(s == 'O')
			{
				crd_o[id_o] = make_float4(x, y, z, 0.f);
				id_o++;
			}
		}
	}

	fclose(fp);
}


__global__ void kernel_prepare(float *q,
                                 int N,
							     float4 *formfactor,
								 float inv_lamda,
								 float inv_distance)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (gid < N)
	{
		float tmp, local_q;
		float fc, fh, fo, fn;

		local_q = FOUR_PI * inv_lamda * sin(0.5 * atan(gid * 0.0732f * inv_distance));
		q[gid]          = local_q;

		tmp = -powf(local_q * 0.25 * INV_PI, 2.0);

		// loop unrolling
		fc = d_atomC[0] * expf(d_atomC[4] * tmp) +
			 d_atomC[1] * expf(d_atomC[5] * tmp) +
			 d_atomC[2] * expf(d_atomC[6] * tmp) +
			 d_atomC[3] * expf(d_atomC[7] * tmp) +
			 d_atomC[8];

		fh = d_atomH[0] * expf(d_atomH[4] * tmp) +
			 d_atomH[1] * expf(d_atomH[5] * tmp) +
			 d_atomH[2] * expf(d_atomH[6] * tmp) +
			 d_atomH[3] * expf(d_atomH[7] * tmp) +
			 d_atomH[8];

		fo = d_atomO[0] * expf(d_atomO[4] * tmp) +
			 d_atomO[1] * expf(d_atomO[5] * tmp) +
			 d_atomO[2] * expf(d_atomO[6] * tmp) +
			 d_atomO[3] * expf(d_atomO[7] * tmp) +
			 d_atomO[8];

		fn = d_atomN[0] * expf(d_atomN[4] * tmp) +
			 d_atomN[1] * expf(d_atomN[5] * tmp) +
			 d_atomN[2] * expf(d_atomN[6] * tmp) +
			 d_atomN[3] * expf(d_atomN[7] * tmp) +
			 d_atomN[8];

		formfactor[gid] = make_float4(fc, fh, fo, fn);
	}
}



int main(int argc, char*argv[])
{
	//-----------------------------------------------------------------------//
	// Read input
	//-----------------------------------------------------------------------//
	lamda = 1.033f; 
	distance = 300.f;
	span = 2048;
	nstreams = 2;

	int opt;
	int fflag = 0;
	extern char   *optarg;  

	// fixme: detect wrong options
	while ( (opt=getopt(argc,argv,"f:l:d:s:n:"))!= EOF) 
	{                    
		switch (opt) {                                                          
			case 'f': 
			         fflag = 1;               // mandatory
			         fname = optarg;                                          
					 break;
			case 'l':
			         lamda = atof(optarg); 
					 break;                                                      
			case 'd': 
			         distance = atof(optarg);
					 break;                                                      
			case 's': 
			         span = atoi(optarg);                             
					 break;                                                      
			case 'n': 
			         nstreams = atoi(optarg);                             
					 break;                                                      
			case '?': 
			         Usage(argv[0]);                                           
					 break;                                                      
			default: 
			         Usage(argv[0]);                                            
					 break;                                                      
		}                                                                       
	}                                                                       

	if (fname == 0) Usage(argv[0]);  

	if (fflag == 0) 
	{
		fprintf(stderr, "%s: missing -f option\n", argv[0]);
        Usage(argv[0]);                                            
	} 

	// check
	std::cout << "file name : " << fname     << std::endl;
	std::cout << "lamda : "     << lamda     << std::endl;
	std::cout << "distance: "   << distance  << std::endl;
	std::cout << "span: "       << span      << std::endl;
	std::cout << "streams: "    << nstreams  << std::endl;

	//-----------------------------------------------------------------------//
	// GPU  
	//-----------------------------------------------------------------------//
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    std::cout << "max texture1d linear: " << deviceProp.maxTexture1DLinear << std::endl;

#if TK
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif




	// unified mem 
	hipMallocManaged((void**)&q,          sizeof(float) * span);
	hipMallocManaged((void**)&formfactor, sizeof(float4) * span);

	// streams
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	for (int i = 0 ; i < nstreams ; i++){
		checkCudaErrors(hipStreamCreate(&(streams[i])));
	}

	dim3 block(256, 1, 1);
	dim3 grid(ceil((float) span / block.x ), 1, 1);


#if TK
	hipEventRecord(start, 0);
#endif

	kernel_prepare <<< grid, block >>> (q, span, formfactor, 1.f/lamda, 1.f/distance);


#if TK
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("runtime = %f ms\n", elapsedTime);
#endif

	readpdb();

	// fixme: one thread doing the i/o
	//        2nd thread working on the gpu prepare kernel


	//-----------------------------------------------------------------------//
	// plan 1 : pair wise compuation
	//-----------------------------------------------------------------------//
	//  cache the crd and atom_type in constant and texture memory
	hipChannelFormatDesc float4Desc = hipCreateChannelDesc<float4>();
	checkCudaErrors(hipBindTexture(NULL, crdc_tex, crd_c, float4Desc));
	checkCudaErrors(hipBindTexture(NULL, crdh_tex, crd_h, float4Desc));
	checkCudaErrors(hipBindTexture(NULL, crdo_tex, crd_o, float4Desc));

	// caculate all the combinations
	// factorial(n) / (factorial(2) * factorial(n-2))
	size_t cc = line_c * (line_c - 1) / 2;
	size_t hh = line_h * (line_h - 1) / 2;
	size_t oo = line_o * (line_o - 1) / 2;
	size_t ch = line_c * line_h;
	size_t co = line_c * line_o;
	size_t ho = line_h * line_o;


	// kernel

	// kernel_cc
	__global void kernel_cc()
	{
		size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

		if (gid < N)	
		{
			for		
			
		}
	}

/*
	hipMemcpyToSymbol(HIP_SYMBOL(atom_type_const), 
                       atom_type, 
					   sizeof(char) * linenum, 
					   0, 
					   hipMemcpyHostToDevice);

	hipChannelFormatDesc float4Desc = hipCreateChannelDesc<float4>();
	checkCudaErrors(hipBindTexture(NULL, crd_tex, crd, float4Desc));
*/

/*
	// slice the workloads for each stream
	int atomNum = linenum; 
	int lastpos = atomNum - 1;

	int step = (atomNum - 1) / nstreams;

	std::vector<int> stream_start;
	std::vector<int> stream_en;

	for(int i = 0; i < nstreams; i++)
	{
		stream_start.push_back(i * step);

		if(i == (nstreams-1))
		{
			stream_end.push_back(atomNum-2);
		}
		else
		{
			stream_end.push_back((i + 1) * step - 1);
		}
	}


	for(int sid = 0; sid < nstreams; sid++)
	{
		kernel_pairwise <<< grid, block, 0, streams[sid] >>> (q, 
                                                              formfactor, 
                                                              stream_start.at(i), 
											                  stream_end.at(i), 
											                  lastpos, 
                                                              span, 
															  sid)
	}
	*/

/*
__global__ void kernel_pairwise(
                                
                                const int start,
                                const int end,
                                const int lastpos,
                                const int N,
                                const int streamID)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;

	if(gid < N)
	{
		for (int startpos = start; startpos <= end; ++startpos) 
		{
			char t1, t2;
			float fj, fk;

			float4 crd_ref = tex1Dfetch(crd_tex, startpos); // load coordinates

			t1 = atom_type_const[startpos];                 // read d_atomtype 1 time, by all N threads

			for(int i = startpos + 1; i <= lastpos; ++i)    // atoms to compare with the base atom
			{
				float4 cur_crd = tex1Dfetch(crd_tex, i);
				float4 distance =  crd_ref - cur_crd;

				t2 = atom_type_const[i];                    // read d_atomtype i times 
				if (t2 == 'C')
				{
					 rzcc_xy = sqrtf(powf(distance.x, 2) + powf(distance.y, 2);
					 rzcc_z  = abs(distance.z);

				}
				else if (t2 == 'H')
				{

				}
				else if (t2 == 'O')
				{
				}
				else
				{}



				iq  += fj_fk * j0(q * );

			} // end of loop
		}
	}// end of if (gid < N)
}
*/

//	for(int i=0; i<linenum; i++)
//		printf("crd[%d] = %f\t%f\t%f\n", i, crd[i].x, crd[i].y, crd[i].z);		




	//std::cout << "element size " << atom_type.size() << std::endl; 

	//-----------------------------------------------------------------------//
	// Free Resource
	//-----------------------------------------------------------------------//
	hipUnbindTexture(crdc_tex);	
	hipUnbindTexture(crdh_tex);	
	hipUnbindTexture(crdo_tex);	

	hipFree(q);
	hipFree(formfactor);
	hipFree(crd_c);
	hipFree(crd_h);
	hipFree(crd_o);
	// hipFree(atom_type);

	

	for (int i = 0 ; i < nstreams ; i++){
		hipStreamDestroy(streams[i]);
	}
	free(streams);

	hipDeviceReset();

	exit (EXIT_SUCCESS);
}
