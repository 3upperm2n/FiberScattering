#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h> // getopt
#include <time.h>

#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_math_constants.h>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h> // float4
#include <hip/hip_runtime_api.h> // check error


#define FOUR_PI (4*HIP_PI_F)
#define INV_PI  (1/HIP_PI_F)
#define TK 1 // time kernel
#define DB 0 // debug 

// texture memory
texture<float4, 1, hipReadModeElementType> crdc_tex;
texture<float4, 1, hipReadModeElementType> crdh_tex;
texture<float4, 1, hipReadModeElementType> crdo_tex;

// constant memory
__device__ __constant__ float 
d_atomC[9]={ 2.31000,  1.02000,  1.58860,  0.865000, 
             20.8439,  10.2075, 0.568700,   51.6512,  
             0.2156};

__device__ __constant__ float 
d_atomH[9]={ 0.493002, 0.322912, 0.140191, 0.040810, 
             10.5109,  26.1257, 3.14236,  57.7997,
			 0.003038};

__device__ __constant__ float 
d_atomO[9]={ 3.04850,  2.28680,  1.54630,  0.867000, 
             13.2771,  5.70110, 0.323900, 32.9089,
			 0.2508};

__device__ __constant__ float 
d_atomN[9]={ 12.2126,  3.13220,  2.01250,  1.16630,  
             0.005700, 9.89330, 28.9975,  0.582600, 
			 -11.52};

__device__ __constant__ float  q_const[2048];
__device__ __constant__ float4 formfactor_const[2048];

//----------------//
// unified memory
//----------------//
float     *q;
float4    *formfactor;
float     *Iq;
float     *Iqz;
float     *Iq_final;
float     *Iqz_final;
float4    *crd_c;
float4    *crd_h;
float4    *crd_o;

//----------------//
// parameters 
//----------------//
char     *fname;
float     lamda;
float     distance;
int       span;
int       nstreams;     // number of cuda streams
int       linenum;
int       line_c;
int       line_h;
int       line_o;

// cuda related
float elapsedTime;
hipEvent_t start, stop;
hipStream_t *streams;

float kernel_runtime = 0.f;

dim3 block(1, 1, 1);
dim3 grid(1, 1, 1);

std::vector<int> beginpos;
std::vector<int> endpos;

int stream_per_com;

void Usage(char *argv0)
{
	const char *help_msg =	
		"\nUsage: %s [options] -f filename\n\n"
		"    -f filename      :file containing atom info\n"		
		"    -l lamda         :angstrom value                 [default=1.033]\n"
		"    -d distance      :specimen to detector distance  [default=300]\n"
		"    -s span          :sampling resolution            [default=2048]\n"
		"    -n nstreams      :number of cuda streams         [default=2]\n";
	fprintf(stderr, help_msg, argv0);
	exit(-1);
}

void readpdb()
{
	// Steps:
	// search the first line start from ATOM
	// the 3rd column, first character is the atom type 
	// 6, 7, 8 column is the x, y, z corordinates

	char line[1000];
	char c1[20];
	char atominfo[20];
	float x, y, z;

	FILE *fp = fopen(fname,"r");
	if(fp == NULL)
		perror("Error opening file!!!\n\n");

	char s;
	linenum = 0;
	line_c  = 0;
	line_h  = 0;
	line_o  = 0;

	while (fgets(line,1000,fp)!=NULL)
	{
		sscanf(line, "%s", c1);
		if(!(strcmp(c1, "ATOM")))
		{
			linenum++;                               // atom list length
			sscanf(line, "%*s %*d %s", atominfo);
			s = atominfo[0];
			if(s == 'C')
				line_c++;
			if(s == 'H')
				line_h++;
			if(s == 'O')
				line_o++;
		}
	}

	rewind(fp);

	std::cout << "line number = " << linenum << std::endl;
	//std::cout << line_c << std::endl;
	//std::cout << line_h << std::endl;
	//std::cout << line_o << std::endl;

	// unified memory
	hipMallocManaged((void**)&crd_c, sizeof(float4) * line_c);
	hipMallocManaged((void**)&crd_h, sizeof(float4) * line_h);
	hipMallocManaged((void**)&crd_o, sizeof(float4) * line_o);

	int id_c, id_h, id_o;
	id_c = id_h = id_o = 0;

	while (fgets(line,1000,fp)!=NULL)
	{
		sscanf(line, "%s", c1);
		if(!(strcmp(c1, "ATOM")))
		{
			sscanf(line, "%*s %*d %s %*s %*d %f %f %f", atominfo, &x, &y, &z);
			s = atominfo[0];
			if(s == 'C')
			{
				crd_c[id_c] = make_float4(x, y, z, 0.f);
				id_c++;
			}

			if(s == 'H')
			{
				crd_h[id_h] = make_float4(x, y, z, 0.f);
				id_h++;	
			}

			if(s == 'O')
			{
				crd_o[id_o] = make_float4(x, y, z, 0.f);
				id_o++;
			}
		}
	}

	fclose(fp);
}


__global__ void kernel_prepare(float *q,
                                 int N,
							     float4 *formfactor,
								 float inv_lamda,
								 float inv_distance)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (gid < N)
	{
		float tmp, local_q;
		float fc, fh, fo, fn;

		local_q = FOUR_PI * inv_lamda * sin(0.5 * atan(gid * 0.0732f * inv_distance));
		q[gid]  = local_q;

		tmp = -powf(local_q * 0.25 * INV_PI, 2.0);

		// loop unrolling
		fc = d_atomC[0] * expf(d_atomC[4] * tmp) +
			 d_atomC[1] * expf(d_atomC[5] * tmp) +
			 d_atomC[2] * expf(d_atomC[6] * tmp) +
			 d_atomC[3] * expf(d_atomC[7] * tmp) +
			 d_atomC[8];

		fh = d_atomH[0] * expf(d_atomH[4] * tmp) +
			 d_atomH[1] * expf(d_atomH[5] * tmp) +
			 d_atomH[2] * expf(d_atomH[6] * tmp) +
			 d_atomH[3] * expf(d_atomH[7] * tmp) +
			 d_atomH[8];

		fo = d_atomO[0] * expf(d_atomO[4] * tmp) +
			 d_atomO[1] * expf(d_atomO[5] * tmp) +
			 d_atomO[2] * expf(d_atomO[6] * tmp) +
			 d_atomO[3] * expf(d_atomO[7] * tmp) +
			 d_atomO[8];

		fn = d_atomN[0] * expf(d_atomN[4] * tmp) +
			 d_atomN[1] * expf(d_atomN[5] * tmp) +
			 d_atomN[2] * expf(d_atomN[6] * tmp) +
			 d_atomN[3] * expf(d_atomN[7] * tmp) +
			 d_atomN[8];

		formfactor[gid] = make_float4(fc, fh, fo, fn);
	}
}

// kernel_cc
__global__ void kernel_cc(int streamID,
                         int line_c,
		                 int N,
		                 int start,
		                 int end,
						 int cc_start,
		                 float* Iq,
		                 float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	int lastpos = line_c - 1;
	float iq, iqz;

	if(gid < N)
	{
		float data = q_const[gid];
		float fj_fk = powf(formfactor_const[gid].x, 2.0); // fj * fk

		// atom list iteration
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdc_tex, j); // crdc_tex

			for(int k = j + 1; k <= lastpos; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdc_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[cc_start + gid  + offset] = iq;
		Iqz[cc_start + gid + offset] = iqz;
	}
}




// kernel_hh
__global__ void kernel_hh(int streamID,
                          int line_h,
		                  int N,
		                  int start,
		                  int end,
						  int hh_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	int lastpos = line_h - 1;
	float iq, iqz;

	if(gid < N)
	{
		float data = q_const[gid];
		float fj_fk = powf(formfactor_const[gid].y, 2.0);  // y = h

		// atom list iteration
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdh_tex, j); // crdh_tex

			for(int k = j + 1; k <= lastpos; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdh_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[hh_start + gid  + offset] = iq;
		Iqz[hh_start + gid + offset] = iqz;
	}
}



// kernel_oo
__global__ void kernel_oo(int streamID,
                          int line_o,
		                  int N,
		                  int start,
		                  int end,
						  int oo_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	int lastpos = line_o - 1;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];
		float fj_fk = powf(formfactor_const[gid].z, 2.0);  // z = o

		// atom list iteration
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdo_tex, j); // crdh_tex

			for(int k = j + 1; k <= lastpos; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdo_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[oo_start  + gid + offset] = iq;
		Iqz[oo_start + gid + offset] = iqz;
	}
}



// kernel_oc: when line_o is longer
__global__ void kernel_oc(int streamID,
                          int len_c,
		                  int N,
		                  int start,
		                  int end,
						  int co_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];

		// x: c		y: h	z: o
		float fj_fk = formfactor_const[gid].z * formfactor_const[gid].x;

		// iterate throught the o atom list 
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdo_tex, j);

			// compare with the c atom list
			for(int k = 0; k < len_c; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdc_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[co_start  + gid + offset] = iq;
		Iqz[co_start + gid + offset] = iqz;
	}
}


// kernel_co: when line_c is longer
__global__ void kernel_co(int streamID,
                          int len_o,
		                  int N,
		                  int start,
		                  int end,
						  int co_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];

		// x: c		y: h	z: o
		float fj_fk = formfactor_const[gid].z * formfactor_const[gid].x;

		// iterate throught the c atom list 
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdc_tex, j);

			// compare with the o atom list
			for(int k = 0; k < len_o; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdo_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[co_start  + gid + offset] = iq;
		Iqz[co_start + gid + offset] = iqz;
	}
}



// kernel_hc: when line_h is longer
__global__ void kernel_hc(int streamID,
                          int len_c,
		                  int N,
		                  int start,
		                  int end,
						  int ch_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];

		// x: c		y: h	z: o
		float fj_fk = formfactor_const[gid].x * formfactor_const[gid].y;

		// iterate throught the h atom list 
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdh_tex, j);

			// compare with the c atom list
			for(int k = 0; k < len_c; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdc_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[ch_start  + gid + offset] = iq;
		Iqz[ch_start + gid + offset] = iqz;
	}
}


// kernel_ch: when line_c is longer
__global__ void kernel_ch(int streamID,
                          int len_h,
		                  int N,
		                  int start,
		                  int end,
						  int ch_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];

		// x: c		y: h	z: o
		float fj_fk = formfactor_const[gid].x * formfactor_const[gid].y;

		// iterate throught the c atom list 
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdc_tex, j);

			// compare with the h atom list
			for(int k = 0; k < len_h; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdh_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[ch_start  + gid + offset] = iq;
		Iqz[ch_start + gid + offset] = iqz;
	}
}



// kernel_ho: when line_h is longer
__global__ void kernel_ho(int streamID,
                          int len_o,
		                  int N,
		                  int start,
		                  int end,
						  int ho_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];
		// x: c		y: h	z: o
		float fj_fk = formfactor_const[gid].y * formfactor_const[gid].z;

		// iterate throught the h atom list 
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdh_tex, j);

			// compare with the o atom list
			for(int k = 0; k < len_o; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdo_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[ho_start  + gid + offset] = iq;
		Iqz[ho_start + gid + offset] = iqz;
	}
}


// kernel_oh: when line_o is longer
__global__ void kernel_oh(int streamID,
                          int len_h,
		                  int N,
		                  int start,
		                  int end,
						  int ho_start,
		                  float* Iq,
		                  float* Iqz)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	int offset = N * streamID;
	float iq, iqz;

	if(gid < N)
	{
		float data  = q_const[gid];
		// x: c		y: h	z: o
		float fj_fk = formfactor_const[gid].y * formfactor_const[gid].z;

		// iterate throught the o atom list 
		for (int j = start; j <= end; ++j) 
		{
			float4 crd_ref = tex1Dfetch(crdo_tex, j);

			// compare with the h atom list
			for(int k = 0; k < len_h; ++k)
			{
				float4 distance = crd_ref - tex1Dfetch(crdh_tex, k);

				iq  += fj_fk * j0(data * sqrt(distance.x * distance.x + distance.y * distance.y));
				iqz += fj_fk * cosf(fabsf(distance.z) * data);
			}
		}

		// accumulate the iterated results 
		Iq[ho_start  + gid + offset] = iq;
		Iqz[ho_start + gid + offset] = iqz;
	}
}




__global__ void kernel_sum(float *Iq,
                           float *Iqz,
                           int    nstreams,
                           int    N,
                           float *Iq_final,
                           float *Iqz_final)
{
	size_t gid = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	// nstreams for each combination  and 6 combinations in total
	int iterations = nstreams;

	if(gid < N)
	{
		float tmp_iq, tmp_iqz;
		tmp_iq = tmp_iqz = 0.f;

		for(int i = 0; i < iterations; i++)
		{
			tmp_iq  +=  Iq[i * N + gid];
			tmp_iqz += Iqz[i * N + gid];
		}
		// accumulate the iterated results 
		Iq_final[gid]  = tmp_iq;
		Iqz_final[gid] = tmp_iqz;
	}
}


void sum_pairwise()
{

#if TK
	hipEventRecord(start, 0);
#endif

	kernel_sum <<< grid, block >>> (Iq, Iqz, nstreams, span, Iq_final, Iqz_final); 

#if TK
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel sum = %f ms\n", elapsedTime);

	kernel_runtime += elapsedTime;
#endif

}

// compute workloads for cc
void work_cc(int streamid)
{
	int len = line_c;
	int sid = streamid % stream_per_com;
	int step = (len - 1) / stream_per_com;

	beginpos.push_back(sid * step);

	if(sid == (stream_per_com-1))
	{
		endpos.push_back(len - 2);
	}
	else
	{
		endpos.push_back((sid + 1) * step - 1);
	}

	//std::cout << "cc sid: "<< sid << std::endl;
}

void work_hh(int streamid)
{
	int len = line_h;
	int sid = streamid % stream_per_com;
	int step = (len - 1) / stream_per_com;

	beginpos.push_back(sid * step);

	if(sid == (stream_per_com-1))
	{
		endpos.push_back(len - 2);
	}
	else
	{
		endpos.push_back((sid + 1) * step - 1);
	}

	//std::cout << "hh sid: "<< sid << std::endl;
}

void work_oo(int streamid)
{
	int len = line_o;
	int sid = streamid % stream_per_com;
	int step = (len - 1) / stream_per_com;

	beginpos.push_back(sid * step);

	if(sid == (stream_per_com-1)){
		endpos.push_back(len - 2);
	}
	else{
		endpos.push_back((sid + 1) * step - 1);
	}
	// std::cout << "oo sid: "<< sid << std::endl;
}

void work_co(int streamid)
{
	int len, step;
	int sid = streamid % stream_per_com;

	if(line_c < line_o)
	{
		len = line_o;		
		step = len / stream_per_com;
		beginpos.push_back(sid * step);
		if(sid == (stream_per_com-1)){
			endpos.push_back(len - 1);
		}else{
			endpos.push_back((sid + 1) * step - 1);
		}
	}
	else
	{
		len = line_c;		
		step = len / stream_per_com;
		beginpos.push_back(sid * step);
		if(sid == (stream_per_com-1)){
			endpos.push_back(len - 1);
		}else{
			endpos.push_back((sid + 1) * step - 1);
		}
	}
	//std::cout << "co sid: "<< sid << std::endl;
}

void work_ch(int streamid)
{
	int len, step;
	int sid = streamid % stream_per_com;

	if(line_c < line_h)
	{
		len = line_h;		
		step = len / stream_per_com;
		beginpos.push_back(sid * step);
		if(sid == (stream_per_com-1)){
			endpos.push_back(len - 1);
		}else{
			endpos.push_back((sid + 1) * step - 1);
		}
	}
	else
	{
		len = line_c;		
		step = len / stream_per_com;
		beginpos.push_back(sid * step);
		if(sid == (stream_per_com-1)){
			endpos.push_back(len - 1);
		}else{
			endpos.push_back((sid + 1) * step - 1);
		}
	}
	//std::cout << "ch sid: "<< sid << std::endl;
}

void work_ho(int streamid)
{
	int len, step;
	int sid = streamid % stream_per_com;

	if(line_o < line_h)
	{
		len = line_h;		
		step = len / stream_per_com;
		beginpos.push_back(sid * step);
		if(sid == (stream_per_com-1)){
			endpos.push_back(len - 1);
		}else{
			endpos.push_back((sid + 1) * step - 1);
		}
	}
	else
	{
		len = line_o;		
		step = len / stream_per_com;
		beginpos.push_back(sid * step);
		if(sid == (stream_per_com-1)){
			endpos.push_back(len - 1);
		}else{
			endpos.push_back((sid + 1) * step - 1);
		}
	}
	//std::cout << "ho sid: "<< sid << std::endl;
}

void run_cc(int i)
{
	kernel_cc <<< grid, block, 0, streams[i] >>> (i, line_c, span, beginpos.at(i), endpos.at(i), 0, Iq, Iqz); 
}

void run_hh(int i)
{
	int hh_start = stream_per_com * span;
	kernel_hh <<< grid, block, 0, streams[i] >>> (i, line_h, span, beginpos.at(i), endpos.at(i), hh_start, Iq, Iqz); 
}

void run_oo(int i)
{
	int oo_start = 2 * stream_per_com * span;
	kernel_oo <<< grid, block, 0, streams[i] >>> (i, line_c, span, beginpos.at(i), endpos.at(i), oo_start, Iq, Iqz); 
}

void run_co(int i)
{
	int co_start = 3 * stream_per_com * span;

	if(line_c < line_o) {
		kernel_oc <<< grid, block, 0, streams[i] >>> (i, line_c, span, beginpos.at(i), endpos.at(i), co_start, Iq, Iqz); 
	}
	else
	{
		kernel_co <<< grid, block, 0, streams[i] >>> (i, line_o, span, beginpos.at(i), endpos.at(i), co_start, Iq, Iqz); 
	}
}

void run_ch(int i)
{
	int ch_start = 4 * stream_per_com * span;

	if(line_c < line_h) {
		kernel_hc <<< grid, block, 0, streams[i] >>> (i, line_c, span, beginpos.at(i), endpos.at(i), ch_start, Iq, Iqz); 
	}
	else
	{
		kernel_ch <<< grid, block, 0, streams[i] >>> (i, line_h, span, beginpos.at(i), endpos.at(i), ch_start, Iq, Iqz); 
	}
}


void run_ho(int i)
{
	int ho_start = 5 * stream_per_com * span;

	if(line_h < line_o) {
		kernel_oh <<< grid, block, 0, streams[i] >>> (i, line_h, span, beginpos.at(i), endpos.at(i), ho_start, Iq, Iqz); 
	}else{
		kernel_ho <<< grid, block, 0, streams[i] >>> (i, line_o, span, beginpos.at(i), endpos.at(i), ho_start, Iq, Iqz); 
	}
}


int main(int argc, char*argv[])
{
	//-----------------------------------------------------------------------//
	// Read input
	//-----------------------------------------------------------------------//
	lamda = 1.033f; 
	distance = 300.f;
	span = 2048;
	nstreams = 30;

	int opt;
	int fflag = 0;
	extern char   *optarg;  

	// fixme: detect wrong options
	while ( (opt=getopt(argc,argv,"f:l:d:s:n:"))!= EOF) 
	{                    
		switch (opt) {                                                          
			case 'f': 
			         fflag = 1;               // mandatory
			         fname = optarg;                                          
					 break;
			case 'l':
			         lamda = atof(optarg); 
					 break;                                                      
			case 'd': 
			         distance = atof(optarg);
					 break;                                                      
			case 's': 
			         span = atoi(optarg);                             
					 break;                                                      
			case 'n': 
			         nstreams = atoi(optarg);                             
					 break;                                                      
			case '?': 
			         Usage(argv[0]);                                           
					 break;                                                      
			default: 
			         Usage(argv[0]);                                            
					 break;                                                      
		}                                                                       
	}                                                                       

	if (fname == 0) Usage(argv[0]);  

	if (fflag == 0) 
	{
		fprintf(stderr, "%s: missing -f option\n", argv[0]);
        Usage(argv[0]);                                            
	} 

	// check
	std::cout << "file name : " << fname     << std::endl;
	std::cout << "lamda : "     << lamda     << std::endl;
	std::cout << "distance: "   << distance  << std::endl;
	std::cout << "span: "       << span      << std::endl;
	std::cout << "streams: "    << nstreams  << std::endl;


	if (nstreams < 6 || (nstreams % 6  != 0))
	{
		std::cout << "nstreams should be multiples of 6 and larger than 5\n";
		exit(EXIT_FAILURE);                                                      
	}

	//-----------------------------------------------------------------------//
	// GPU  
	//-----------------------------------------------------------------------//
    //int dev = 0;
    //hipSetDevice(dev);
    //hipDeviceProp_t deviceProp;
    //hipGetDeviceProperties(&deviceProp, dev);
    //printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    //std::cout << "max texture1d linear: " << deviceProp.maxTexture1DLinear << std::endl;
	// set device                                                               
	hipDeviceProp_t device_prop;                                                 
	int dev_id = findCudaDevice(argc, (const char **) argv);                    
	checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));             

	if (!device_prop.managedMemory) {                                           
		fprintf(stderr, "Unified Memory not supported on this device\n");       
		hipDeviceReset();                                                      
		exit(EXIT_FAILURE);                                                      
	}                                                                           

	if (device_prop.computeMode == hipComputeModeExclusive || device_prop.computeMode == hipComputeModeProhibited)
	{                                                                           
		fprintf(stderr, "This sample requires a device in either default or process exclusive mode\n");
		hipDeviceReset();                                                      
		exit(EXIT_FAILURE);                                                      
	}      




#if TK
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif


	// unified mem 
	checkCudaErrors(hipMallocManaged((void**)&q,          sizeof(float)  * span));
	checkCudaErrors(hipMallocManaged((void**)&formfactor, sizeof(float4) * span));

	// for each combination launch nstreams
	checkCudaErrors(hipMallocManaged((void**)&Iq,  sizeof(float) * span * nstreams));
	checkCudaErrors(hipMallocManaged((void**)&Iqz, sizeof(float) * span * nstreams));

	// streams
	streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
	for (int i = 0 ; i < nstreams ; i++){
		checkCudaErrors(hipStreamCreate(&(streams[i])));
	}

	// configure the kernel grid size
	block.x = 256;
	grid.x  = ceil( (float) span / block.x );


#if TK
	hipEventRecord(start, 0);
#endif

	kernel_prepare <<< grid, block >>> (q, span, formfactor, 1.f/lamda, 1.f/distance);

#if TK
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel prepare = %f ms\n", elapsedTime);

	kernel_runtime += elapsedTime;
#endif

	// copy q and formfactor to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(q_const),                   q,  sizeof(float) * span, 0, hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(formfactor_const), formfactor, sizeof(float4) * span, 0, hipMemcpyDeviceToDevice);

	// fixme: one thread doing the i/o
	//        2nd thread working on the gpu prepare kernel
	readpdb();


	//-----------------------------------------------------------------------//
	// plan 1 : pair wise compuation
	//-----------------------------------------------------------------------//
	//  cache the crd and atom_type in constant and texture memory
	hipChannelFormatDesc float4Desc = hipCreateChannelDesc<float4>();
	checkCudaErrors(hipBindTexture(NULL, crdc_tex, crd_c, float4Desc));
	checkCudaErrors(hipBindTexture(NULL, crdh_tex, crd_h, float4Desc));
	checkCudaErrors(hipBindTexture(NULL, crdo_tex, crd_o, float4Desc));

	// output offsets
	//   cc : 0
	//   hh : 1 * nstreams * span
	//   oo : 2 * nstreams * span
	//   co : 3 * nstreams * span
	//   ch : 4 * nstreams * span
	//   ho : 5 * nstreams * span


	stream_per_com =  nstreams / 6; 


	//std::cout << "stream_per_com : "<< stream_per_com << std::endl;
	std::cout << "line_c : "<< line_c << std::endl;
	std::cout << "line_h : "<< line_h << std::endl;
	std::cout << "line_o : "<< line_o << std::endl;

	//------------------------------------------//
	// assign the workloads
	//------------------------------------------//
	for(int i = 0; i < nstreams; i++)
	{
		if(i< stream_per_com)
		{
			work_cc(i);
		}else if (i < 2 * stream_per_com){
			work_hh(i);
		}else if (i < 3 * stream_per_com){
			work_oo(i);
		}else if (i < 4 * stream_per_com){
			work_co(i);
		}else if (i < 5 * stream_per_com){
			work_ch(i);
		}else {
			work_ho(i);
		}
	}                                      

/*
	std::cout << "size of beginpos : " << beginpos.size() << std::endl;
//	std::cout << "size of endpos : "   << endpos.size()   << std::endl;
	for(int i=0; i<beginpos.size(); i++)
	{
		std::cout << beginpos[i] << " - "<< endpos[i] << std::endl;
	}
*/

#if TK
	hipEventRecord(start, 0);
#endif

	// when line_h is longer
	for(int i = 0; i < nstreams; i++)
	{
		if(i< stream_per_com)
		{
			run_cc(i);	
		}
		else if (i < 2 * stream_per_com)
		{
			run_hh(i);	
		}
		else if (i < 3 * stream_per_com)
		{
			run_oo(i);	
		}
		else if (i < 4 * stream_per_com)
		{
			run_co(i);	
		}
		else if (i < 5 * stream_per_com)
		{
			run_ch(i);	
		}
		else 
		{
			run_ho(i);	
		}

	}                                      

#if TK
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("kernel pair-wise = %f ms\n", elapsedTime);

	kernel_runtime += elapsedTime;
#endif

	//-----------------------------------------------------------------------//
	// sum pair wise compuation
	//-----------------------------------------------------------------------//
	checkCudaErrors(hipMallocManaged((void**)&Iq_final,  sizeof(float) * span));
	checkCudaErrors(hipMallocManaged((void**)&Iqz_final, sizeof(float) * span));

	sum_pairwise();

	//std::cout << span << std::endl;

	hipDeviceSynchronize(); 


//	for(int i = 0; i < span; i++){
//		// printf("Iq[%d] = %f\n", i, Iq_final[i]);		
//	}

	std::cout << "kernels execution time = " << kernel_runtime << " ms\n";


	//-----------------------------------------------------------------------//
	// Free Resource
	//-----------------------------------------------------------------------//
	hipUnbindTexture(crdc_tex);	
	hipUnbindTexture(crdh_tex);	
	hipUnbindTexture(crdo_tex);	

	hipFree(q);
	hipFree(formfactor);
	hipFree(crd_c);
	hipFree(crd_h);
	hipFree(crd_o);
	hipFree(Iq);
	hipFree(Iqz);
	hipFree(Iq_final);
	hipFree(Iqz_final);

	for (int i = 0 ; i < nstreams ; i++){
		hipStreamDestroy(streams[i]);
	}
	free(streams);

	hipDeviceReset();

	exit (EXIT_SUCCESS);
}
